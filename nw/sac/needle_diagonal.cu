#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <omp.h>


#define LIMIT -999

#ifndef SIZE
#define SIZE 4096
#endif

#define BLOCK 256
#define BLOCK_X 16
#define BLOCK_Y 16

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
inline __device__ int maximum( int a, int b, int c)
{
  int m = a;
  
  if( m > b ) m = b;
  if( m > c ) m = c; 

  return( m);
}


int blosum62[24][24] = {
{ 4, -1, -2, -2,  0, -1, -1,  0, -2, -1, -1, -1, -1, -2, -1,  1,  0, -3, -2,  0, -2, -1,  0, -4},
{-1,  5,  0, -2, -3,  1,  0, -2,  0, -3, -2,  2, -1, -3, -2, -1, -1, -3, -2, -3, -1,  0, -1, -4},
{-2,  0,  6,  1, -3,  0,  0,  0,  1, -3, -3,  0, -2, -3, -2,  1,  0, -4, -2, -3,  3,  0, -1, -4},
{-2, -2,  1,  6, -3,  0,  2, -1, -1, -3, -4, -1, -3, -3, -1,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{ 0, -3, -3, -3,  9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4},
{-1,  1,  0,  0, -3,  5,  2, -2,  0, -3, -2,  1,  0, -3, -1,  0, -1, -2, -1, -2,  0,  3, -1, -4},
{-1,  0,  0,  2, -4,  2,  5, -2,  0, -3, -3,  1, -2, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -2,  0, -1, -3, -2, -2,  6, -2, -4, -4, -2, -3, -3, -2,  0, -2, -2, -3, -3, -1, -2, -1, -4},
{-2,  0,  1, -1, -3,  0,  0, -2,  8, -3, -3, -1, -2, -1, -2, -1, -2, -2,  2, -3,  0,  0, -1, -4},
{-1, -3, -3, -3, -1, -3, -3, -4, -3,  4,  2, -3,  1,  0, -3, -2, -1, -3, -1,  3, -3, -3, -1, -4},
{-1, -2, -3, -4, -1, -2, -3, -4, -3,  2,  4, -2,  2,  0, -3, -2, -1, -2, -1,  1, -4, -3, -1, -4},
{-1,  2,  0, -1, -3,  1,  1, -2, -1, -3, -2,  5, -1, -3, -1,  0, -1, -3, -2, -2,  0,  1, -1, -4},
{-1, -1, -2, -3, -1,  0, -2, -3, -2,  1,  2, -1,  5,  0, -2, -1, -1, -1, -1,  1, -3, -1, -1, -4},
{-2, -3, -3, -3, -2, -3, -3, -3, -1,  0,  0, -3,  0,  6, -4, -2, -2,  1,  3, -1, -3, -3, -1, -4},
{-1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1, -2, -4,  7, -1, -1, -4, -3, -2, -2, -1, -2, -4},
{ 1, -1,  1,  0, -1,  0,  0,  0, -1, -2, -2,  0, -1, -2, -1,  4,  1, -3, -2, -2,  0,  0,  0, -4},
{ 0, -1,  0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1,  1,  5, -2, -2,  0, -1, -1,  0, -4},
{-3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3, -1,  1, -4, -3, -2, 11,  2, -3, -4, -3, -2, -4},
{-2, -2, -2, -3, -2, -1, -2, -3,  2, -1, -1, -2, -1,  3, -3, -2, -2,  2,  7, -1, -3, -2, -1, -4},
{ 0, -3, -3, -3, -1, -2, -2, -3, -3,  3,  1, -2,  1, -1, -2, -2,  0, -3, -1,  4, -3, -2, -1, -4},
{-2, -1,  3,  4, -3,  0,  1, -1,  0, -3, -4,  0, -3, -3, -2,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{-1,  0,  0,  1, -3,  3,  4, -2,  0, -3, -3,  1, -1, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2,  0,  0, -2, -1, -1, -1, -1, -1, -4},
{-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1}
};


__global__ void upper_left(int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty)
{
   int idx, r, c;

   idx = blockIdx.x*blockDim.x+threadIdx.x; 

   if( idx >= i) return;

   r = i - idx;  
   c = i + 1 - r;  

   int base = r*max_cols+c;
   input_itemsets[base] 
		= maximum( input_itemsets[base-max_cols-1]+ reference[base], 
			   input_itemsets[base-1] - penalty, 
			   input_itemsets[base-max_cols] - penalty);
}

__global__ void lower_right(int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty)
{
   int idx, r, c;
  
   idx = blockIdx.x*blockDim.x+threadIdx.x; 

   if( idx >= i) return;

   r = max_rows-1-idx; 
   c = max_cols-i+idx;  
   

   int base = r*max_cols+c;
   input_itemsets[base] 
		= maximum( input_itemsets[base-max_cols-1]+ reference[base], 
			   input_itemsets[base-1] - penalty, 
			   input_itemsets[base-max_cols] - penalty);
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
  int max_rows, max_cols, penalty;
  int *input_itemsets, *reference, *input_itemsets_d, *reference_d;
  int i,j;
  struct timeval tv1, tv2;
  double runtime;

  penalty = 10;
    
  // the lengths of the two sequences should be able to divided by 16.
  // And at current stage  max_rows needs to equal max_cols
  max_rows = SIZE + 1;
  max_cols = SIZE + 1;


  reference = (int *)malloc( max_rows * max_cols * sizeof(int) );
  input_itemsets = (int *)malloc( max_rows * max_cols * sizeof(int) );
  hipMalloc((void**)&input_itemsets_d, max_rows*max_cols*sizeof(int));
  hipMalloc((void**)&reference_d, max_rows*max_cols*sizeof(int));
	
  srand (7);

  for (i = 0 ; i < max_cols; i++) {
    for (j = 0 ; j < max_rows; j++) {
      input_itemsets[i*max_cols+j] = rand()%SIZE;
    }
  }

  srand (2012);

  for (i = 0 ; i < max_cols; i++) {
    for (j = 0 ; j < max_rows; j++) {
      reference[i*max_cols+j] = rand()%20;
    }
  }


  hipMemcpy(input_itemsets_d, input_itemsets, sizeof(int)*max_rows*max_cols, hipMemcpyHostToDevice); 
  hipMemcpy(reference_d, reference, sizeof(int)*max_rows*max_cols, hipMemcpyHostToDevice); 


  gettimeofday( &tv1, NULL);

  for( i = 1; i < max_cols; i++) {
/*
    input_itemsets = with {
                       ( [1,1] <= iv=[r,c] < [1+i,1+i]) {
                         if( r == (i - c + 1)) {
                           res = maximum( input_itemsets[r-1, c-1]+ reference[r, c], 
                                          input_itemsets[r, c-1] - penalty, 
			                  input_itemsets[r-1, c] - penalty);
                         }
                         else {
                           res = input_itemsets[r,c];
                         }
                       }:res;
                     }:modarray( input_itemsets);
*/
    dim3 block(BLOCK);
    dim3 grid(i/BLOCK+1);
    upper_left<<<grid, block>>>( input_itemsets_d, reference_d, max_rows, max_cols, i, penalty);
  }


  for( i = max_cols-2; i >= 1; i--) {
/*
    input_itemsets = with {
                       ( [1+i,1+i] <= iv=[r,c] < [max_rows,max_cols]) {
                         if( r == (max_cols - c + i)) { 
                           res = maximum( input_itemsets[r-1, c-1]+ reference[r, c], 
                                          input_itemsets[r, c-1] - penalty, 
			                  input_itemsets[r-1, c] - penalty);
                         }
                         else {
                           res = input_itemsets[r,c];
                         }
                       }:res;
                     }:modarray( input_itemsets);
*/
    dim3 block(BLOCK);
    dim3 grid(i/BLOCK+1);
    lower_right<<<grid, block>>>( input_itemsets_d, reference_d, max_rows, max_cols, i, penalty);
  }

  hipDeviceSynchronize();

  gettimeofday( &tv2, NULL);
  runtime = ((tv2.tv_sec*1000.0+ tv2.tv_usec/1000.0)-(tv1.tv_sec*1000.0+ tv1.tv_usec/1000.0));
  printf("Runtime: %f\n", runtime);

  hipMemcpy(input_itemsets, input_itemsets_d, sizeof(int)*max_rows*max_cols, hipMemcpyDeviceToHost); 


#ifdef OUTPUT 
  for( i = 0; i < max_rows; i++) {
    for( j = 0; j < max_cols; j++) {
      printf("%d ", input_itemsets[i*max_cols+j]);
    }
    printf("\n");
  }
  return( 0);
#else
  printf("%d\n", input_itemsets[0]);
  return(0);
#endif
}


